#include "hip/hip_runtime.h"
#include "Adapt_Mesh.cuh"

gpuDevice device;

deviceFunction
void write_signal_in_bounding_box::operator()(const uint64_t global_id) const {
	// Write function
};


hostFunction
int32_t setInitialParticles(
	const floatType* input_signal_dvc,
	Particle* output_active_nodes_dvc,
	const cartesianMesh& signal_bounding_box,
	const cartesianMesh& signal_domain
) {

	// Create the signal in the bounding box. Initialized to 0
	const uint64_t size_input_signal = signal_domain.total_nodes();
	const uint64_t size_signal_in_bounding_box = signal_bounding_box.total_nodes();

	// Create and fill with 0 the signal_in_bounding_box array (remember to free memory afterwards):
	floatType* signal_in_bounding_box_dvc = nullptr;
	try {
		gpu_device.device_malloc((void**)&signal_in_bounding_box_dvc, size_signal_in_bounding_box * sizeof(floatType));
		gpu_device.device_memSet(signal_in_bounding_box_dvc, 0, size_signal_in_bounding_box * sizeof(floatType));

		//Fill the signal_in_bounding_box_dvc
		uint16_t threads = fmin(THREADS_P_BLK, size_signal_in_bounding_box);
		uint64_t blocks = floor((size_signal_in_bounding_box - 1) / threads) + 1;

		gpu_device.launch_kernel(blocks, threads, write_signal_in_bounding_box{
			   input_signal_dvc,
			   signal_in_bounding_box_dvc,
			   signal_domain,
			   signal_bounding_box,
			   size_signal_in_bounding_box
			}
		);

		gpu_device.device_free(signal_in_bounding_box);
	}
	catch (const std::exception& e) {
		std::cout << "Caught exception: " << e.what() << std::endl;
		if (signal_in_bounding_box_dvc) {
			gpu_device.device_free(signal_in_bounding_box_dvc);
		}
		return EXIT_FAILURE;
	}

	// Create amr_handle
	waveletTransform amr_engine;

	amr_engine.set_min_refinement_level(0);
	amr_engine.set_max_refinement_level(log2(signal_bounding_box.nodes_per_dim()));
	amr_engine.set_initial_signal_dvc2dvc(signal_in_bounding_box_dvc);

	amr_engine.compute_wavelet_transform();
	get_detail_above_threshold_nodes(amr_engine, output_active_nodes_dvc, signal_bounding_box);

	//return amr_engine.transformed_signal();
	return EXIT_SUCCESS;
};

hostFunction
void get_detail_above_threshold_nodes(waveletTransform& amr_engine, Particle* particle_locations, const cartesianMesh& signal_domain) {

	// We get the number of selected nodes because we'll read the first nr_selected_nodes indeces in the bounding box mesh
	uint32_t nr_selected_nodes = amr_engine.sorted_assigned_nodes();


	const uint32_t Threads = fmin(THREADS_P_BLK, nr_selected_nodes);
	const uint64_t Blocks = floor((nr_selected_nodes - 1) / Threads) + 1;

	device.launch_kernel(Blocks, Threads,
		get_nodes_from_indeces<ELEMENTS_AT_A_TIME>{
			particle_locations,
			signal_domain,
			amr_engine.assigned_node_indeces_dvc(),
			nr_selected_nodes
		}
	);
};