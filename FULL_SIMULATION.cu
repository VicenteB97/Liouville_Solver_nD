#include "hip/hip_runtime.h"
#include "Classes.cuh"
#include "Simulation_parameters.cuh"
#include "Dynamics.cuh"
#include "Parameter_distributions.cuh"
#include "Adapt_Mesh.cuh"
#include "Mat_Ops.cuh"
#include "Impulse_transformations.cuh"

#include "Debugging.cuh"

using namespace thrust::placeholders; // this is useful for the multiplication of a device vector by a constant

__host__ int PDF_ITERATIONS(std::vector<double>* store_PDFs,
							const std::vector<Param_vec>* Parameter_Mesh,
							const gridPoint* H_Mesh,
							thrust::host_vector<double>* H_PDF,
							const int& LvlFine,
							const int& LvlCoarse,
							const int& PtsPerDim,
							const int& Grid_Nodes,
							const std::vector<Time_Impulse_vec>	time_vector,
							const float& deltaT,
							const int& ReinitSteps);

__host__ int _SL_PDF_ITERATIONS(std::vector<double>* store_PDFs,
								const std::vector<Param_vec>* Parameter_Mesh,
								const gridPoint* H_Mesh,
								thrust::host_vector<double>* H_PDF,
								const int& LvlFine,
								const int& LvlCoarse,
								const int& PtsPerDim,
								const int& Grid_Nodes,
								const std::vector<Time_Impulse_vec>	time_vector,
								const float& deltaT,
								const int& ReinitSteps);

//--------------------------------------------------------------------------------------------- //

/// <summary>
/// This function computes the evolution of an initial PDF (defined in "Parameter_distributions.cuh")
///  according to the dynamics parameters (also defined in "Parameter_distributions.cuh"), with some 
/// given simulation parameters (given in "Simulation_parameters.cuh"). 
/// </summary>
/// <returns> An integer (0 if there are no problems)</returns>
int PDF_EVOLUTION() {
	std::cout << "You must choose: \n - FINEST / COARSEST MESH LEVEL \n - If you wish to include IMPULSE terms \n - FINAL time and TIMESTEP";
	std::cout << "\n - Reinitialization Steps \n - SAMPLES PER PARAMETER \n - BACKWARD or FORWARD Lagrangian ADVECTION method?\n";
	std::cout << "(REMINDER) You can type \"-1\" to  exit the program while selecting the simulation parameters.\n";
	// Pre-Simulation Data initialization -------------------------------------------------------------------------------

	// ----------------------------------------------------------------------------------------------- //
	// ---------------------------------- OBTAIN INFO FROM TERMINAL ---------------------------------- //
	// ----------------------------------------------------------------------------------------------- //
					int LvlFine, LvlCoarse = 0;
					std::cout << "Finest level in the domain?: ";
					std::cin >> LvlFine;
					if (LvlFine == -1){
						std::cout << "Exiting simulation\n";
						return -1;
					}

					std::cout << "Coarsest level in the domain?: ";
					std::cin >> LvlCoarse;
					if (LvlCoarse == -1){
						std::cout << "Exiting simulation\n";
						return -1;
					}
	// ----------------------------------------------------------------------------------------------- //
	// ----------------------------------------------------------------------------------------------- //
	// ----------------------------------------------------------------------------------------------- //

	const int PtsPerDim  = (int)powf(2, LvlFine);
	const int Grid_Nodes = (int)powf(PtsPerDim, DIMENSIONS);

	// Build H_Mesh here (TO BE MODIFIED FOR OTHER DIMENSIONS)
	const gridPoint Domain_Center = {0.5, 0.5};
	const gridPoint Domain_Radius = {12, 12};

	gridPoint* H_Mesh = new gridPoint[Grid_Nodes]; // maybe it could be implemented as constant memory?

	// GENERAL DIMENSION Cartesian coordinate grid
	for (unsigned int i = 0; i < Grid_Nodes; i++){
		for (unsigned int d = 0; d < DIMENSIONS; d++){
			unsigned int j = floor(positive_rem(i, pow(PtsPerDim, d + 1))/pow(PtsPerDim, d));
			H_Mesh[i].dim[d] = ((double) j / (PtsPerDim - 1) - Domain_Center.dim[d]) * Domain_Radius.dim[d]; 
		}
	}

	// Time simulation data Definition: -------------------------------------------------------------------------------
	int			ReinitSteps;
	double		deltaT;
	std::vector<Time_Impulse_vec> time_vector;

	bool impulse = true;
	
	// ----------------------------------------------------------------------------------------------- //
	// ---------------------------------- OBTAIN INFO FROM TERMINAL ---------------------------------- //
	// ----------------------------------------------------------------------------------------------- //
					char ans;
					std::cout << "Include impulse terms? (Y=Yes, N=No): ";
					std::cin >> ans;
					while (ans != 'Y' && ans != 'y') {
						if (ans == 'N' || ans == 'n') { impulse = false; break; }
						else if(ans == '-1'){
						std::cout << "Exiting simulation\n";
							return -1;
						}
						else {
							std::cout << "SYNTAX Error. Include impulse terms? (Y=Yes, N=No): ";
							std::cin >> ans;
						}
					}
	// ----------------------------------------------------------------------------------------------- //
	// ----------------------------------------------------------------------------------------------- //
	// ----------------------------------------------------------------------------------------------- //

	int aux_result = Simul_Data_Def(time_vector, deltaT, ReinitSteps, impulse);
	if (aux_result == -1){
		std::cout << "Exiting simulation.\n";
		return aux_result;
	}

	// --------------------------------------------------------------------------------------------
	// --------------------------------------------------------------------------------------------
	// ---------------------- PDF EVOLUTION STEPS AND SIMULATION RUN ------------------------------
	// --------------------------------------------------------------------------------------------
	// 	   Note: If we want to do some kind of optimzation procedure regarding the Liouville Eq.,
	// 	   you have to make a function such as PDF_EVOLUTION but you must modify THIS following part
	// 	   PDF_EVOLUTION simply computes the evolution of a PDF according to some sim. parameters
	// 	   as well as the corresponding dynamics parameters.
	// --------------------------------------------------------------------------------------------
	// --------------------------------------------------------------------------------------------
	// 1.- PARAMETER H_MESH biuld up
	int n_samples[PARAM_DIMENSIONS];						// number of samples per parameter
	
	// ----------------------------------------------------------------------------------------------- //
	// ---------------------------------- OBTAIN INFO FROM TERMINAL ---------------------------------- //
	// ----------------------------------------------------------------------------------------------- //
				for (int k = 0; k < PARAM_DIMENSIONS; k++) {
					std::cout << "How many samples for parameter " << k + 1 << " ? ";
					std::cin >> n_samples[k];
					while (n_samples[k] == 0){ 
						std::cout << "At least 1 sample must be selected. How many samples for parameter " << k + 1 << " ? "; 
						std::cin >> n_samples[k];
					}
					if (n_samples[k] == -1){
					std::cout << "Exiting simulation.\n"; return -1;}
				}
	// ----------------------------------------------------------------------------------------------- //
	// ----------------------------------------------------------------------------------------------- //
	// ----------------------------------------------------------------------------------------------- //
	int Random_Samples = 1;
	for (unsigned int i = 0; i < PARAM_DIMENSIONS; i++){
		Random_Samples *= n_samples[i];
	}

	std::vector<Param_vec>	Parameter_Mesh(Random_Samples);					// Full parameter array
	double 					Dist_Params[PARAM_DIMENSIONS * 2];
	char 					Dist_Names[PARAM_DIMENSIONS];

	// 1st RV mean and variance
	Dist_Names[0]  = 'N';
	Dist_Params[0] = 0.2;
	Dist_Params[1] = 0.02;

	// 2nd RV mean and variance
	Dist_Names[1]  = 'N';
	Dist_Params[2] = 3;
	Dist_Params[3] = 0.3;

	RANDOMIZE(n_samples, Random_Samples, &Parameter_Mesh, Dist_Params, Dist_Names);	

	std::cout << "Total number of random samples: " << Random_Samples << ".\n";

	// --------------------------------------------------------------------------------------------
	// --------------------------------------------------------------------------------------------

	// --------------------------------------------------------------------------------------------
	// --------------------------------------------------------------------------------------------
	// 2.- INITIAL PDF build up
	thrust::host_vector<double> H_PDF(Grid_Nodes);	 // PDF values at the fixed, high-res grid (CPU)

	double IC_dist_params[DIMENSIONS * 2];

	// one pair per dimension
	IC_dist_params[0] = 1.75;  // mean
	IC_dist_params[1] = sqrt(0.025); // var

	IC_dist_params[2] = 0.00;  // mean
	IC_dist_params[3] = sqrt(0.025); // var

	PDF_INITIAL_CONDITION(PtsPerDim, H_Mesh, H_PDF, IC_dist_params); // initialize the grid and the PDF at the grid nodes (change so as to change the parameters as well)

// --------------------------------------------------------------------------------------------
// --------------------------------------------------------------------------------------------

// --------------------------------------------------------------------------------------------
// --------------------------------------------------------------------------------------------
// 3.- Evolution simulation
	std::cout << "What kind of Lagrangian advection? (\"1\" = Backward, \"2\" = Forward): ";
	char number;
	std::cin >> number;

	bool _SL_simulation = false;

	// ----------------------------------------------------------------------------------------------- //
	// ---------------------------------- OBTAIN INFO FROM TERMINAL ---------------------------------- //
	// ----------------------------------------------------------------------------------------------- //
			while (number != '2') {
				if(number == '1'){
					_SL_simulation = true;
					break;
				}
				else if(number == '-1'){
				std::cout << "Exiting simulation\n";
					return -1;
					break;
				}
				else {
					std::cout << "SYNTAX ERROR.\nWhat kind of Lagrangian advection? (\"1\" = Backward, \"2\" = Forward): ";
					std::cin >> number;
				}
			}
	// ----------------------------------------------------------------------------------------------- //
	// ----------------------------------------------------------------------------------------------- //
	// ----------------------------------------------------------------------------------------------- //

	std::vector<double>	store_PDFs(0);		 // H_PDF storage for post-processing

	auto start = std::chrono::high_resolution_clock::now();

	if (_SL_simulation) {
		// GOOD:
		// - More stable under large time-steps
		// - Simpler idea + implementation
		// - (Slightly) faster in low dimensions and samples ((<2 - 18)%)
		// - Great interpolation schemes available (for dimensions 1, 2 and maybe 3)

		// BAD:
		// - Not possible to make it much faster
		// - No nice scaling for larger dimensions and samples
		// - Undefined behavior in critical points
		std::cout << "Simulating with BACKWARD advection\n";
		aux_result = _SL_PDF_ITERATIONS(&store_PDFs, &Parameter_Mesh, H_Mesh, &H_PDF, LvlFine, LvlCoarse, PtsPerDim, Grid_Nodes, time_vector, deltaT, ReinitSteps);
		// Conservative method + nd-linear interpolation?
		if (aux_result == -1){
			std::cout << "An error has occured. Exiting simulation.\n";
			return aux_result;
	}
	}
	else {
		// GOOD:
		// - Easier to scale to higher dimensions/samples
		// - (Can be) more GPU-friendly
		// - Several ideas to make it much better
		//		- Tree structure for particle search
		//		- Conservative scheme is done by sparse matrix multiplication
		// - Ideas are closer to reality
		// - Handles critical points naturally

		// BAD:
		// - More steps involved per timestep
		// - Re-interpolation every few steps is a must (time-step is very limited)
		// - Interpolation must be of RBF-type
		// - Not as stable under large timesteps
		std::cout << "Simulating with FORWARD advection\n";
		aux_result = PDF_ITERATIONS(&store_PDFs, &Parameter_Mesh, H_Mesh, &H_PDF, LvlFine, LvlCoarse, PtsPerDim, Grid_Nodes, time_vector, deltaT, ReinitSteps);
		// To do: CREATE TREE STRUCT FOR THE AMR-POINTS + conservative scheme
		if (aux_result == -1){
			std::cout << "An error has occured. Exiting simulation.\n";
			return aux_result;
		}
	}

	auto end = std::chrono::high_resolution_clock::now();
	std::chrono::duration<float> duration = end - start; // duration

// --------------------------------------------------------------------------------------------
// --------------------------------------------------------------------------------------------
// --------------------------------------------------------------------------------------------
// ---------------------- STORAGE INTO COMPUTER HDD for post-processing -----------------------
// --------------------------------------------------------------------------------------------
// --------------------------------------------------------------------------------------------
// --------------------------------------------------------------------------------------------
	std::cout << "Simulation time: " << duration.count() << " seconds. Memory to be written: " << (float)store_PDFs.size() * sizeof(double) / 1000000 << " MB. Write? (Y=Yes, N=no)\n";
	std::cin >> ans;

	if ((ans == 'Y') || (ans == 'y')) {
		std::cout << "Writing to memory...please wait (it may take over 10 seconds)\n";

		// Simulation Information
		std::ofstream file1;
		file1.open("Simulation_Info.csv");
		if (file1.is_open()) {
			//file1 << "Total Grid Points," << "Points per dimension," << "Grid X min," << "Grid X max," << "Grid Y min," << "Grid Y max," << "Time values," << "Simulation cost" << "t0" << "deltaT" << "Reinitialization Steps" << "\n";
			file1 << Grid_Nodes << "," << PtsPerDim << "," << H_Mesh[0].dim[0] << "," << H_Mesh[Grid_Nodes - 1].dim[0]
				<< "," << H_Mesh[0].dim[1] << "," << H_Mesh[Grid_Nodes - 1].dim[1] << "," << time_vector.size() << "," << duration.count() << "," << "\n";

			for (int i = 0; i < time_vector.size() - 1; i++) {
				file1 << time_vector[i].time << ",";
			}
			file1 << time_vector[time_vector.size() - 1].time;
			file1.close();
		}
		else {
			std::cout << "Fail.\n";
		}

		// Simulation Data
		std::ofstream myfile;
		myfile.open("Mean_PDFs.csv");
		if (myfile.is_open()) {
			for (int k = 0; k < store_PDFs.size() - 1; k++) {
				myfile << store_PDFs[k] << ",";
			}
			myfile << store_PDFs[store_PDFs.size() - 1];
			myfile.close();
			std::cout << "Completed!\n";
		}
		else {
			std::cout << "Failed!!\n";
		}
	}

	delete[] H_Mesh;

	return 0;
}


//---------------------------------------------------------------------------------------------
//---------------------------------------------------------------------------------------------
//---------------------------------------------------------------------------------------------
//------------------------ MAIN COMPUTATION FUNCTIONS -----------------------------------------
//---------------------------------------------------------------------------------------------
// 1.- Runge-Kutta integrator (particle + value)   --------------------------------------------
// 2.- Iterations of the Liouville-Equation solver --------------------------------------------
//---------------------------------------------------------------------------------------------

/// <summary>
/// This function computes the advection of the particles created by AMR.
/// It uses the RK4 scheme for the spatial variables advection and the Simpson rule for the exponential integral.
/// </summary>
/// <param name="H_Mesh"> - Particle location (spatial variables)</param>
/// <param name="PDF"> - PDF value at the corresponding particle location </param>
/// <param name="parameters"> - Parameters to be used for the vector field and its corresponding divergence function</param>
/// <param name="t0"> - Inital time for starting the simulation</param>
/// <param name="deltaT"> - time step used in the simulation</param>
/// <param name="ReinitSteps"> - Number of steps before needing a re-interpolation</param>
/// <param name="Adapt_Points"> - Number of particles as computed by the AMR scheme</param>
/// <param name="Random_Samples"> - Number of random parameter samples</param>
/// <returns></returns>
__global__ void RungeKutta(	gridPoint* 			H_Mesh,
							double* 			PDF,
							const Param_vec* 	parameters,
							double				t0,
							const double		deltaT,
							const int			ReinitSteps,
							const int			Adapt_Points,
							const int			Random_Samples) {

	const int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < Adapt_Points * Random_Samples) {
		int steps = 0;

		// So, the total amount of advections are going to be: (n� particles x n� of samples)
		const int i_sample = floorf(i / Adapt_Points);

		// AUXILIARY DATA TO RUN THE ITERATIONS
		gridPoint k0, k1, k2, k3, aux;
		double	  Int1, Int2, Int3;

		gridPoint x0 	= H_Mesh[i]; 	// register storing the initial particle dim
		double Int_PDF 	= PDF[i];	// register storing the initial particle value

		while (steps < ReinitSteps) {

			// Particle flow
			k0 = VECTOR_FIELD(x0, t0, parameters[i_sample]);

			aux = Mult_by_Scalar(deltaT / 2, k0);
			k1 = VECTOR_FIELD(x0 + aux, t0 + deltaT / 2, parameters[i_sample]);

			aux = Mult_by_Scalar(deltaT / 2, k1);
			k2 = VECTOR_FIELD(x0 + aux, t0 + deltaT / 2, parameters[i_sample]);

			aux = Mult_by_Scalar(deltaT, k2);
			k3 = VECTOR_FIELD(x0 + aux, t0 + deltaT, parameters[i_sample]);

			k1 = Mult_by_Scalar(2, k1);
			k2 = Mult_by_Scalar(2, k2);

			aux = x0 + Mult_by_Scalar(deltaT / 6, (k0 + k3 + k1 + k2)); // New particle dim

			// Integration of PDF
			Int1 = DIVERGENCE_FIELD(x0, t0, parameters[i_sample]);

			x0 	 = Mult_by_Scalar(0.5, (x0 + aux));
			Int2 = DIVERGENCE_FIELD(x0, (2 * t0 + deltaT) / 2, parameters[i_sample]);

			Int3 = DIVERGENCE_FIELD(aux, t0 + deltaT, parameters[i_sample]);

			Int_PDF *= exp(-deltaT / 6 * (Int1 + 4 * Int2 + Int3)); // New particle value (change for the Hermite interpolation in the midpoint)

			// Reinit step
			x0 = aux;
			t0 += deltaT;
			steps++;
		}

		H_Mesh[i] 	= aux;
		PDF[i]		= Int_PDF;
	}
}


/// <summary>
/// 
/// This function computes the Liouville Eq. iterations from t0 to tF. Each iteration consists of the following steps:
/// 1st) Compute the AMR of the initial PDF. 
/// 2nd) Create as many particle families as random samples there are.
/// 3rd) Advect all particles via RungeKutta function. 
/// 4th) Interpolate all families of particles onto the underlying high-res. fixed grid. 
/// 5th) Compute ensemble mean.
/// 6th) Store final PDF into the iteration vector for further post-processing and/or evolution visualization.
/// 
/// </summary>
/// <param name="store_PDFs"> - Iteration storage (all PDF evolution "slides")</param>
/// <param name="Parameter_Mesh"> - Parameter values H_Mesh + joint PDF value </param>
/// <param name="H_Mesh"> - Initial, fixed, high-res. grid where we obtain the adapted particles</param>
/// <param name="H_PDF"> - Initial PDF values at the H_Mesh</param> 
/// <param name="LvlFine"> - Finest level in the AMR grids</param>
/// <param name="LvlCoarse"> - Coarsest level in the AMR grids</param>
/// <param name="PtsPerDim"> - Points per dimension</param>
/// <param name="Grid_Nodes"> - Total number of grid nodes</param>
/// <param name="time_vector"> - vector containing all the info about the time and implulses in the simulation</param>
/// <param name="deltaT"> - timestep for the RK4 scheme</param>
/// <param name="ReinitSteps"> - deltaT steps until next re-interpolation in high res. grid</param>
/// <returns></returns>
__host__ int PDF_ITERATIONS(std::vector<double>* store_PDFs,
							const std::vector<Param_vec>* Parameter_Mesh,
							const gridPoint* H_Mesh,
							thrust::host_vector<double>* H_PDF,
							const int& LvlFine,
							const int& LvlCoarse,
							const int& PtsPerDim,
							const int& Grid_Nodes,
							const std::vector<Time_Impulse_vec> time_vector,
							const float& deltaT,
							const int& ReinitSteps) {

//--------------------------------------------------------------------------------------------//
//--------------------------------------------------------------------------------------------//
//--------------------------------------------------------------------------------------------//
//------------------- INITIALIZATION OF VECTORS/ARRAYS THAT WILL BE USED ---------------------//
//--------------------------------------------------------------------------------------------//
//--------------------------------------------------------------------------------------------//
//--------------------------------------------------------------------------------------------//
	std::vector<gridPoint>	AdaptGrid(0);			// Particle positions to be used for simulation (corresponding dim from AMR)
	std::vector<double>		AdaptPDF(0);			// PDF value at the particle positions (corresponding values from AMR)
	std::vector<gridPoint>	Full_AdaptGrid(0);		// Final adapted grid (adapted grid x number of samples)
	std::vector<double>		Full_AdaptPDF(0);		// Final adapted PDF (adapted grid x number of samples)

	thrust::device_vector<gridPoint>	GPU_Part_Position;		// Particle positions (for the GPU)
	thrust::device_vector<double>		GPU_AdaptPDF;			// PDF value at Particle positions (for the GPU)
	thrust::device_vector<Param_vec>	GPU_Parameter_Mesh;		// Parameter H_Mesh array (for the GPU)

	const int Random_Samples = Parameter_Mesh->size();			// number of random parameter samples

	thrust::device_vector<gridPoint>	GPU_Mesh(H_Mesh, H_Mesh + Grid_Nodes);	// Fixed Grid Nodes (for the GPU)
	thrust::device_vector<double>		GPU_PDF = *H_PDF;						// PDF values at fixed Grid Nodes (for the GPU)

	// auxiliary variable that will be used for ensemble mean computation
	double Sum_Rand_Params = 0;
	for (int i = 0; i < Random_Samples; i++) {
		Sum_Rand_Params += Parameter_Mesh->at(i).Joint_PDF;
	}


// ------------------ DEFINITION OF THE INTERPOLATION VARIABLES AND ARRAYS ------------------ //
	int Adapt_Points, Total_Particles, MaxNeighborNum;

	const double disc_X = (H_Mesh[1].dim[0] - H_Mesh[0].dim[0]);	// H_Mesh discretization size
	const double search_radius = 4.75 * disc_X;						// max radius to search ([4,6] appears to be optimal)

	const int	 max_steps = 1000;		 // max steps at the Conjugate Gradient (CG) algorithm
	const double in_tolerance = pow(10, -8); // CG stop tolerance

	thrust::device_vector<int>		GPU_Index_array;
	thrust::device_vector<double>	GPU_Mat_entries;
	thrust::device_vector<int>		GPU_Num_Neighbors;

// --------------------------------------------------------------------------------------------
// --------------------------------------------------------------------------------------------
// --------------------------------------------------------------------------------------------
// ---------------------------------- SIMULATION TIME!! ---------------------------------------
// --------------------------------------------------------------------------------------------
// --------------------------------------------------------------------------------------------
// --------------------------------------------------------------------------------------------

// -------------------- Store the 1st PDF (Init. Cond.) -----------------------------------
	store_PDFs->insert(store_PDFs->end(), H_PDF->begin(), H_PDF->end());

	// ------------------------------------------------------------------------------------
	int j = 0;

	while (j < time_vector.size() - 1) {

		auto start_2 = std::chrono::high_resolution_clock::now();

		float	t0 = time_vector[j].time,
			tF = time_vector[j + 1].time;

		std::cout << "/-------------------------------------------------------------------/\n";
		if (t0 == tF) {
			std::cout << "RVT transformation at time: " << t0 << "\n";
		}
		else {
			std::cout << "Simulation time: " << t0 << " to " << tF << "\n";
		}

// 1.- Initial step Adaptive H_Mesh Refinement. First store the initial PDF with AMR performed
		ADAPT_MESH_REFINEMENT_nD(*H_PDF, &AdaptPDF, H_Mesh, &AdaptGrid, LvlFine, LvlCoarse, PtsPerDim);

	// 1.1.- COMPUTE THE TRANSFORMATION OF THE PDF (IF THERE IS ONE)
		if (time_vector[j].impulse && (time_vector[j].samples[0] > 0 || time_vector[j].samples[1] > 0)) {
			// compute transformaton
			//std::cout << "Computing impulse transformation.\n";

			int success_impulse = IMPULSE_TRANSFORM_PDF(H_Mesh, &AdaptGrid, H_PDF, &AdaptPDF, time_vector[j], Grid_Nodes, PtsPerDim);

			if (success_impulse != 0) {
				//std::cout << "Something went wrong...\n";
				std::cin.get();
				return success_impulse;
			}
			else {
				//std::cout << "Transformation done...continuing with Liouville PDE\n";

				AdaptGrid.clear();
				AdaptPDF.clear();

			}
		}
	// 1.2.- Error?
		else if (time_vector[j].impulse) {
			std::cout << "Check you impulse samples...you need at least one sample for the impulse.\n";
			break;
		}
	// 1.3.- COMPUTE THE SMOOTH EVOLUTION VIA LIOUVILLE GIBBS / CONTINUITY EQUATION
		else {

			Adapt_Points = AdaptGrid.size();
			Total_Particles = Adapt_Points * Random_Samples;

			Full_AdaptGrid.clear();
			Full_AdaptPDF.clear();

			// 1.2.- Append the optimal particles once per sample!
			for (int k = 0; k < Random_Samples; k++) {
				Full_AdaptGrid.insert(Full_AdaptGrid.end(), AdaptGrid.begin(), AdaptGrid.end());
				Full_AdaptPDF.insert(Full_AdaptPDF.end(), AdaptPDF.begin(), AdaptPDF.end());
			}

			// Upload information to the GPU
			GPU_AdaptPDF 		= Full_AdaptPDF;
			GPU_Part_Position 	= Full_AdaptGrid;
			GPU_Parameter_Mesh 	= *Parameter_Mesh;

			std::cout << "Size of relevant PDF points (per sample): " << Adapt_Points << "\n";	// this allows to check if the info is passed to the GPU correctly

			// Clear CPU info about the AMR procedure: no longer needed
			AdaptGrid.clear();
			AdaptPDF.clear();

			MaxNeighborNum = (int)fminf(200, Adapt_Points);				// maximum neighbors to search


// ------------------ RESIZING OF THE INTERPOLATION MATRIX ------------------ //
			GPU_Index_array.resize(MaxNeighborNum * Total_Particles);
			GPU_Mat_entries.resize(MaxNeighborNum * Total_Particles);
			GPU_Num_Neighbors.resize(Total_Particles);
// -------------------------------------------------------------------------- //
			// Determine threads and blocks for the simulation
			int Threads = (int)fminf(THREADS_P_BLK, Total_Particles);
			int Blocks  = (int)floorf((Total_Particles - 1) / Threads) + 1;

// ------------------------------------------------------------------------------------ //
// -------------------------- POINT ADVECTION ----------------------------------------- //
// ------------------------------------------------------------------------------------ //
			// Using RK4 for time integration of characteristic curves
			RungeKutta << <Blocks, Threads >> >(raw_pointer_cast(&GPU_Part_Position[0]), 
												raw_pointer_cast(&GPU_AdaptPDF[0]),
												raw_pointer_cast(&GPU_Parameter_Mesh[0]), 
												t0, 
												deltaT, 
												ReinitSteps, 
												Adapt_Points, 
												Random_Samples);
			gpuError_Check(hipDeviceSynchronize()); // Here, the entire H_Mesh points (those that were selected) and PDF points (same) have been updated.

// ----------------------------------------------------------------------------------- //
// -------------------------- INTERPOLATION ------------------------------------------ //
// ----------------------------------------------------------------------------------- //
	// 1.- Build Matix in GPU (indexes, dists and neighbors) Using Exahustive search...
			Exh_PP_Search << <Blocks, Threads >> > (raw_pointer_cast(&GPU_Part_Position[0]), 
													raw_pointer_cast(&GPU_Part_Position[0]), 
													raw_pointer_cast(&GPU_Index_array[0]),
													raw_pointer_cast(&GPU_Mat_entries[0]), 
													raw_pointer_cast(&GPU_Num_Neighbors[0]), 
													MaxNeighborNum, 
													Adapt_Points, 
													Total_Particles, 
													search_radius);
			gpuError_Check(hipDeviceSynchronize());

	// 2.- Iterative solution (Conjugate Gradient) to obtain coefficients of the RBFs
			thrust::device_vector<double>	GPU_lambdas(Total_Particles);	// solution vector (RBF weights)
			thrust::fill(GPU_lambdas.begin(), GPU_lambdas.end(), 0);		// this will serve as the initial condition

			int err = CONJUGATE_GRADIENT_SOLVE<double>( GPU_lambdas, 
														GPU_Index_array, 
														GPU_Mat_entries, 
														GPU_Num_Neighbors, 
														GPU_AdaptPDF, 
														Total_Particles, 
														MaxNeighborNum, 
														max_steps, 
														in_tolerance);
			if (err == -1) { return err; }

	// 3.- Multiplication of matrix-lambdas to obtain new points
			bool new_restart_mthd = true;

			if (new_restart_mthd) {
			// Re-define Threads and Blocks
			Threads = fminf(THREADS_P_BLK, Total_Particles);
			Blocks  = floorf((Grid_Nodes - 1) / Threads) + 1;

			thrust::fill(GPU_PDF.begin(), GPU_PDF.end(), 0);	// PDF is reset to 0, so that we may use atomic adding

			// I'M GOING TO FIND THE NEAREST GRID NODES TO EACH PARTICLE
			// TO DO: MAKE IT USING BATCHES! (MAYBE BATCHES OF 10-50) BECAUSE IT'S NOT WORKING PROPERLY! IT'S THE ATOMICADD FUNCTION'S FAULT
			RESTART_GRID_FIND_GN<<< Blocks, Threads >>>(raw_pointer_cast(&GPU_Part_Position[0]),
														raw_pointer_cast(&GPU_PDF[0]),
														raw_pointer_cast(&GPU_lambdas[0]),
														raw_pointer_cast(&GPU_Mesh[0]),
														raw_pointer_cast(&GPU_Parameter_Mesh[0]),
														search_radius,
														H_Mesh[0],
														disc_X,
														PtsPerDim,
														Adapt_Points,
														Total_Particles);
			gpuError_Check(hipDeviceSynchronize());

			// Correct any possible negative PDF values
			CORRECTION<<<Blocks, Threads>>>(raw_pointer_cast(&GPU_PDF[0]), Grid_Nodes);
			gpuError_Check(hipDeviceSynchronize());
				
			}
			else {
				// Re-define Threads and Blocks
				Threads = fminf(THREADS_P_BLK, Grid_Nodes);
				Blocks = floorf((Grid_Nodes - 1) / Threads) + 1;

				auto start_2 = std::chrono::high_resolution_clock::now();

				RESTART_GRID<<< Blocks, Threads >>>(raw_pointer_cast(&GPU_PDF[0]), 
													raw_pointer_cast(&GPU_Mesh[0]), 
													raw_pointer_cast(&GPU_Part_Position[0]),
													raw_pointer_cast(&GPU_lambdas[0]), 
													raw_pointer_cast(&GPU_Parameter_Mesh[0]), 
													search_radius, 
													Grid_Nodes, 
													Adapt_Points, 
													Total_Particles);
				gpuError_Check(hipDeviceSynchronize());

				auto end_2 = std::chrono::high_resolution_clock::now();

				std::chrono::duration<float> duration_2 = end_2 - start_2;
			}

			thrust::transform(GPU_PDF.begin(), GPU_PDF.end(), GPU_PDF.begin(), 1 / Sum_Rand_Params * _1); // we use the thrust::placeholders here (@ the last input argument)

			*H_PDF = GPU_PDF; // Send back to CPU
		}

		// Store info in cumulative variable
		store_PDFs->insert(store_PDFs->end(), H_PDF->begin(), H_PDF->end());
		j++;

		auto end_2 = std::chrono::high_resolution_clock::now();

		std::chrono::duration<float> duration_2 = end_2 - start_2;
		std::cout << "Liouville iteration took " << duration_2.count() << " seconds\n";
		std::cout << "/-------------------------------------------------------------------/\n";
	}
	return 0;
}


/// <summary>
/// Semi-Lagrangian Runge Kutta method.
/// </summary>
/// <param name="H_Mesh"></param>
/// <param name="PDF"></param>
/// <param name="Adapt_Mesh"></param>
/// <param name="lambdas"></param>
/// <param name="search_radius"></param>
/// <param name="parameters"></param>
/// <param name="tF"></param>
/// <param name="deltaT"></param>
/// <param name="ReinitSteps"></param>
/// <param name="Grid_Nodes"></param>
/// <param name="Adapt_Points"></param>
/// <param name="Random_Samples"></param>
/// <returns></returns>
__global__ void _SL_RungeKutta( const gridPoint* 	H_Mesh,
								double* 			PDF,
								const gridPoint* 	Adapt_Mesh,
								const double* 		lambdas,
								const double		search_radius,
								const Param_vec* 	parameters,
								double				t0,
								const double		deltaT,
								const int			ReinitSteps,
								const int			Grid_Nodes,
								const int			Adapt_Points,
								const int			Random_Samples) {

	const int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < Grid_Nodes) {

		double New_PDF_val 	= 0;			// cumulative variable for the new PDF value
		// double discretization_length = H_Mesh[1].dim[0] - H_Mesh[0].dim[0];

		for (int i_sample = 0; i_sample < Random_Samples; i_sample++) {

			gridPoint	 x0 					= H_Mesh[i];
			unsigned int steps 					= 0;
			Param_vec	 parameter_sample 		= parameters[i_sample];
			double		 aux_inverse_Integral 	= 1;

			while (steps < ReinitSteps) {
				// AUXILIARY DATA TO RUN THE ITERATIONS
				gridPoint k0, k1, k2, k3, aux, x_MP = x0;

				// Particle INVERSE flow
				k0 = VECTOR_FIELD(x0, t0 + deltaT, parameter_sample);
				k0 = Mult_by_Scalar(-1, k0);

				aux = Mult_by_Scalar(deltaT / 2, k0);
				k1  = VECTOR_FIELD(x0 + aux, t0 + deltaT / 2, parameter_sample);
				k1  = Mult_by_Scalar(-1, k1);

				aux = Mult_by_Scalar(deltaT / 2, k1);
				k2  = VECTOR_FIELD(x0 + aux, t0 + deltaT / 2, parameter_sample);
				k2  = Mult_by_Scalar(-1, k2);

				aux = Mult_by_Scalar(deltaT, k2);
				k3  = VECTOR_FIELD(x0 + aux, t0, parameter_sample);

				k1 = Mult_by_Scalar(2, k1);
				k2 = Mult_by_Scalar(2, k2);

				x0 = x0 + Mult_by_Scalar(deltaT / 6, (k0 + k1 + k2 - k3));	// New particle dim WHY IS IT "-"?

				// Inverse integrate the imagined value for the PDF at the characteristic curve
				// we are going to try and do it twice, comparing the interpolated value with the transported value and adjusting
				double Int1 = DIVERGENCE_FIELD(x0, t0 + deltaT, parameter_sample);

				x_MP = Mult_by_Scalar(0.5, x_MP + x0);
				double Int2 = DIVERGENCE_FIELD(x_MP, t0 + (deltaT) / 2, parameter_sample);

				double Int3 = DIVERGENCE_FIELD(x_MP, t0, parameter_sample);

				aux_inverse_Integral *= exp(-deltaT / 6 * (Int1 + 4 * Int2 + Int3)); // New particle value (change for the Hermite interpolation in the midpoint)

				// Reinit step
				t0 -= deltaT;
				steps++;
			}

		// // compute the nearest node from the discretization
		// int nearest_node = round(x0.dim[0] / discretization_length);
		// for (unsigned int d = 0; d < DIMENSIONS; d++){
		// 	nearest_node += round(x0.dim[d] / discretization_length) * pow(Grid_Nodes, d);
		// }

		// double Old_PDF = 0;
		// for (unsigned int l = 0; l < (int)pow((int)2 * ceil(search_radius / discretization_length), DIMENSIONS); l++){
		// 	// go back to the "lowest" node:
		// 	for (unsigned int d = 0; d < DIMENSIONS; d++){
		// 		nearest_node -= round(search_radius / discretization_length) * pow(Grid_Nodes, d);
		// 	}

		// 	// now, nearest node is the lowest node in the grid. We have to go up until we cover the whole tile
		// }

			// find nearby grid nodes and interpolate the "upstream" point
		// // compute the nearest node from the discretization
		// 	int nearest_node = round(x0.dim[0] / discretization_length);
		// 	for (unsigned int d = 1; d < DIMENSIONS; d++){
		// 		nearest_node += round(x0.dim[d] / discretization_length) * pow(Grid_Nodes, d);
		// 	}

			double Old_PDF = 0;
			for (unsigned int j = 0; j < Adapt_Points; j++) {
				double dist = Distance(x0, Adapt_Mesh[j]) / search_radius;
				if (dist <= 1) {
					Old_PDF += RBF(search_radius, dist) * lambdas[j];
				}
			}

			// Integration of PDF
			New_PDF_val += Old_PDF * aux_inverse_Integral * parameter_sample.Joint_PDF;
		}
		PDF[i] = cuda_fmax(New_PDF_val, 0);
	}
}


/// <summary>
/// This function computes the Liouville equation evolution via a Semi-Lagrangian iterations. The procedure is the following: 
/// 1st) Perform the AMR of the Initial Condition PDF
/// 2nd) Inverse-advect all particles starting at the grid nodes at "t0 + deltaT"
/// 3.- Obtain the PDF value at the new starting point
/// 4.- Forward-advect the PDF value to the new grid node
/// </summary>
/// <param name="store_PDFs"></param>
/// <param name="Parameter_Mesh"></param>
/// <param name="H_Mesh"></param>
/// <param name="H_PDF"></param>
/// <param name="LvlFine"></param>
/// <param name="LvlCoarse"></param>
/// <param name="PtsPerDim"></param>
/// <param name="Grid_Nodes"></param>
/// <param name="time_vector"></param>
/// <param name="deltaT"></param>
/// <param name="ReinitSteps"></param>
/// <returns></returns>
__host__ int _SL_PDF_ITERATIONS(std::vector<double>* store_PDFs,
	const std::vector<Param_vec>* Parameter_Mesh,
	const gridPoint* H_Mesh,
	thrust::host_vector<double>* H_PDF,
	const int& LvlFine,
	const int& LvlCoarse,
	const int& PtsPerDim,
	const int& Grid_Nodes,
	const std::vector<Time_Impulse_vec>	time_vector,
	const float& deltaT,
	const int& ReinitSteps) {

	//--------------------------------------------------------------------------------------------//
	//--------------------------------------------------------------------------------------------//
	//--------------------------------------------------------------------------------------------//
	//------------------- INITIALIZATION OF VECTORS/ARRAYS THAT WILL BE USED ---------------------//
	//--------------------------------------------------------------------------------------------//
	//--------------------------------------------------------------------------------------------//
	//--------------------------------------------------------------------------------------------//
	std::vector<gridPoint>	AdaptGrid;			// Particle positions to be used for simulation (corresponding dim from AMR)
	std::vector<double>		AdaptPDF;			// PDF value at the particle positions (corresponding values from AMR)

	thrust::device_vector<gridPoint>	GPU_Part_Position;		// Particle positions (for the GPU)
	thrust::device_vector<double>		GPU_AdaptPDF;			// PDF value at Particle positions (for the GPU)

	thrust::device_vector<Param_vec>	GPU_Parameter_Mesh = *Parameter_Mesh;		// Parameter H_Mesh array (for the GPU)

	const int Random_Samples = Parameter_Mesh->size();			// number of random parameter samples

	thrust::device_vector<gridPoint>	GPU_Mesh(H_Mesh, H_Mesh + Grid_Nodes);	// Fixed Grid Nodes (for the GPU)
	thrust::device_vector<double>		GPU_PDF(Grid_Nodes);					// PDF values at fixed Grid Nodes (for the GPU)

	// auxiliary variable that will be used for ensemble mean computation
	double Sum_Rand_Params = 0;
	for (int i = 0; i < Random_Samples; i++) {
		Sum_Rand_Params += Parameter_Mesh->at(i).Joint_PDF;
	}


	// ------------------ DEFINITION OF THE INTERPOLATION VARIABLES AND ARRAYS ------------------ //
	int Adapt_Points, MaxNeighborNum;

	double Iteration_information[2];

	const double disc_X 		= (H_Mesh[1].dim[0] - H_Mesh[0].dim[0]);	// H_Mesh discretization size
	const double search_radius  = 4.5 * disc_X;								// max radius to search ([6,8] appears to be optimal)

	const int	 max_steps 	  	= 1000;				// max steps at the Conjugate Gradient (CG) algorithm
	const double in_tolerance 	= pow(10, -8);  	// CG stop tolerance

	thrust::device_vector<int>		GPU_Index_array;
	thrust::device_vector<double>	GPU_Mat_entries;
	thrust::device_vector<int>		GPU_Num_Neighbors;

	// ------------------ AUXILIARIES FOR THE INTEPROLATION PROC. ------------------------------- //
	thrust::device_vector<double>	GPU_lambdas;	// solution vector
	thrust::device_vector<double>	GPU_R;			// residual vector
	thrust::device_vector<double>	GPU_AUX;		// auxiliary vector for computation storage
	thrust::device_vector<double>	GPU_AP;			// vector for storing the A*P multiplication
	thrust::device_vector<double>	GPU_P;			// P vector

// --------------------------------------------------------------------------------------------
// --------------------------------------------------------------------------------------------
// --------------------------------------------------------------------------------------------
// ---------------------------------- SIMULATION TIME!! ---------------------------------------
// --------------------------------------------------------------------------------------------
// --------------------------------------------------------------------------------------------
// --------------------------------------------------------------------------------------------

// -------------------- Store the 1st PDF (Init. Cond.) -----------------------------------
	store_PDFs->insert(store_PDFs->end(), H_PDF->begin(), H_PDF->end());

	// ------------------------------------------------------------------------------------
	int j = 0;
	int Blocks, Threads;

	while (j < time_vector.size() - 1) {


		auto start_2 = std::chrono::high_resolution_clock::now();

		float	t0 = time_vector[j].time,
			tF = time_vector[j + 1].time;

		std::cout << "/-------------------------------------------------------------------/\n";

		// 1.- Initial step Adaptive H_Mesh Refinement. First store the initial PDF with AMR performed
		ADAPT_MESH_REFINEMENT_nD(*H_PDF, &AdaptPDF, H_Mesh, &AdaptGrid, LvlFine, LvlCoarse, PtsPerDim);

		Adapt_Points = AdaptGrid.size();

		if (time_vector[j].impulse && (time_vector[j].samples[0] > 0 || time_vector[j].samples[1] > 0)) {
			// compute transformaton
			//std::cout << "Computing impulse transformation.\n";
			std::cout << "RVT transformation at time: " << t0 << "\n";

			std::cout << "Number of points: " << Adapt_Points << "\n";
			int success_impulse = IMPULSE_TRANSFORM_PDF(H_Mesh, &AdaptGrid, H_PDF, &AdaptPDF, time_vector[j], Grid_Nodes, PtsPerDim);

			if (success_impulse != 0) {
				std::cout << "Something went wrong...\n";
				std::cin.get();
				return success_impulse;
			}
			else {
				//std::cout << "Transformation done...continuing with Liouville PDE\n";

				AdaptGrid.clear();
				AdaptPDF.clear();

			}
		}
		// 1.2.- Error?
		else if (time_vector[j].impulse) {
			std::cout << "Check you impulse samples...you need at least one sample for the impulse.\n";
			break;
		}
		// 1.3.- COMPUTE THE SMOOTH EVOLUTION VIA LIOUVILLE GIBBS / CONTINUITY EQUATION
		else {
			// 2.- Perform RBF interpolation

			std::cout << "Simulation time: " << t0 << " to " << tF << "\n";

			std::cout << "Number of points: " << Adapt_Points << "\n";

			GPU_Part_Position.resize(Adapt_Points);
			GPU_AdaptPDF.resize(Adapt_Points);

			GPU_Part_Position = AdaptGrid;
			GPU_AdaptPDF = AdaptPDF;
			MaxNeighborNum = (int)powf(2 * search_radius / disc_X, DIMENSIONS);

			// Don't need them anymore
			AdaptGrid.clear();
			AdaptPDF. clear();

		// ------------------ RESIZING OF THE INTERPOLATION MATRIX ------------------ //
			GPU_Index_array.resize(MaxNeighborNum * Adapt_Points);
			GPU_Mat_entries.resize(MaxNeighborNum * Adapt_Points);
			GPU_Num_Neighbors.resize(Adapt_Points);

		// ------------------ RESIZING FOR THE INTEPROLATION PROC. ------------------//
			// Lambdas: coefficients of the RBFs
			GPU_lambdas.resize(Adapt_Points);
			thrust::fill(GPU_lambdas.begin(), GPU_lambdas.end(), 0);	// this will serve as the initial condition

			// Auxiliary vectors for the Conjugate Gradient
			GPU_R.resize(Adapt_Points);
			GPU_AUX.resize(Adapt_Points);
			GPU_AP.resize(Adapt_Points);
			GPU_P.resize(Adapt_Points);

			// Determine threads and blocks for the simulation
			Threads = (int)fminf(THREADS_P_BLK, Adapt_Points);
			Blocks = (int)floorf(Adapt_Points / Threads) + 1;

			//Search for nearby points
			Exh_PP_Search << <Blocks, Threads >> > (raw_pointer_cast(&GPU_Part_Position[0]), raw_pointer_cast(&GPU_Part_Position[0]), raw_pointer_cast(&GPU_Index_array[0]),
				raw_pointer_cast(&GPU_Mat_entries[0]), raw_pointer_cast(&GPU_Num_Neighbors[0]), MaxNeighborNum, Adapt_Points, Adapt_Points, search_radius);

			gpuError_Check(hipDeviceSynchronize());
			//std::cout << "Point Search: done\n";

		// Actually interpolate:
			// 2.- Iterative solution (Conjugate Gradient) to obtain coefficients of the RBFs
			int k = 1;

			// Auxiliary vectors and values
			double sq_error;		// square error
			bool   flag = true;		// to stop the iterations

		// Initialize Conjugate gradient method ----------------------------------------------------
				// Compute A * X0
			MATRIX_VECTOR_MULTIPLICATION << < Blocks, Threads >> > (raw_pointer_cast(&GPU_AUX[0]), raw_pointer_cast(&GPU_lambdas[0]), raw_pointer_cast(&GPU_Index_array[0]),
				raw_pointer_cast(&GPU_Mat_entries[0]), Adapt_Points, raw_pointer_cast(&GPU_Num_Neighbors[0]), MaxNeighborNum);

			gpuError_Check(hipDeviceSynchronize());

			// Compute R=B-A*X0
			DIFF_VECS << <Blocks, Threads >> > (raw_pointer_cast(&GPU_R[0]), raw_pointer_cast(&GPU_AdaptPDF[0]), raw_pointer_cast(&GPU_AUX[0]), Adapt_Points);
			gpuError_Check(hipDeviceSynchronize());

			double Alpha, R0_norm, aux, beta;

			GPU_P = GPU_R;

			while (flag) { // this flag is useful to know when we have arrived to the desired tolerance
			// Alpha computation (EVERYTHING IS CORRECT!)
				// 1.1.- Compute AP=A*P
				MATRIX_VECTOR_MULTIPLICATION << < Blocks, Threads >> > (raw_pointer_cast(&GPU_AP[0]), raw_pointer_cast(&GPU_P[0]), raw_pointer_cast(&GPU_Index_array[0]),
					raw_pointer_cast(&GPU_Mat_entries[0]), Adapt_Points, raw_pointer_cast(&GPU_Num_Neighbors[0]), MaxNeighborNum);

				gpuError_Check(hipDeviceSynchronize());

				// 1.2.- Compute P'*AP
				thrust::transform(GPU_P.begin(), GPU_P.end(), GPU_AP.begin(), GPU_AUX.begin(), thrust::multiplies<double>());
				aux = thrust::reduce(thrust::device, GPU_AUX.begin(), GPU_AUX.end());

				// 1.3.- R'*R
				thrust::transform(GPU_R.begin(), GPU_R.end(), GPU_R.begin(), GPU_AUX.begin(), thrust::multiplies<double>());
				R0_norm = thrust::reduce(thrust::device, GPU_AUX.begin(), GPU_AUX.end());

				Alpha = R0_norm / aux;

				// New X and R: (new, old, scalar, driving vec, total length)
					// 1.- Update Lambdas
				UPDATE_VEC << <Blocks, Threads >> > (raw_pointer_cast(&GPU_lambdas[0]), raw_pointer_cast(&GPU_lambdas[0]), Alpha, raw_pointer_cast(&GPU_P[0]), Adapt_Points);

				gpuError_Check(hipDeviceSynchronize());

				// 2.- Update residuals 
				UPDATE_VEC << <Blocks, Threads >> > (raw_pointer_cast(&GPU_R[0]), raw_pointer_cast(&GPU_R[0]), -Alpha, raw_pointer_cast(&GPU_AP[0]), Adapt_Points);

				gpuError_Check(hipDeviceSynchronize());

				// Compute residual norm
				thrust::transform(GPU_R.begin(), GPU_R.end(), GPU_R.begin(), GPU_AUX.begin(), thrust::multiplies<double>());
				sq_error = thrust::reduce(thrust::device, GPU_AUX.begin(), GPU_AUX.end()); // sum of its elements
				sq_error = sqrt(sq_error);

				if (sq_error < in_tolerance) {
					Iteration_information[0] = (int)k;
					Iteration_information[1] = sq_error;
					//std::cout << "Convergence success. Iterations:" << Iteration_information[0] << " Error:" << Iteration_information[1] << "\n";
					flag = false;
				}
				else if (k > max_steps) {
					std::cout << "No convergence was obtained. Last residual norm was: " << sq_error << "\n";
					std::cout << "/-------------------------------------------------------------------/\n";

					std::cin.get();
					return 0;
				}
				else {
					beta = sq_error * sq_error / R0_norm;

					UPDATE_VEC << <Blocks, Threads >> > (raw_pointer_cast(&GPU_P[0]), raw_pointer_cast(&GPU_R[0]), beta, raw_pointer_cast(&GPU_P[0]), Adapt_Points);

					gpuError_Check(hipDeviceSynchronize());
					k++;
				}
			}

			//double aux_res = thrust::reduce(thrust::device, GPU_lambdas.begin(), GPU_lambdas.end());

			// 3.- NOW, WE HAVE TO DO THE INVERSE FLOW VIA THE SEMI-LAGRANGIAN ADVECTION

		// ------------------------------------------------------------------------------------ //
		// --------------------------- POINT ADVECTION ---------------------------------------- //
		// ------------------------------------------------------------------------------------ //
		// Using RK4 for time integration of characteristic curves
			Threads = (int)fminf(THREADS_P_BLK, Grid_Nodes);
			Blocks = (int)floorf(Grid_Nodes / Threads) + 1;

			// make batches HERE: IDEALLY, IT WILL OPTIMIZE THE TIME SPENT IN GPU 
			_SL_RungeKutta << <Blocks, Threads >> > (raw_pointer_cast(&GPU_Mesh[0]), raw_pointer_cast(&GPU_PDF[0]), raw_pointer_cast(&GPU_Part_Position[0]),
				raw_pointer_cast(&GPU_lambdas[0]), search_radius, raw_pointer_cast(&GPU_Parameter_Mesh[0]), t0, deltaT, ReinitSteps, Grid_Nodes, Adapt_Points, Random_Samples);
			gpuError_Check(hipDeviceSynchronize()); // Here, the entire H_Mesh points (those that were selected) and PDF points (same) have been updated.
			//std::cout << "Advection: done\n";

			thrust::transform(GPU_PDF.begin(), GPU_PDF.end(), GPU_PDF.begin(), 1 / Sum_Rand_Params * _1); // we use the thrust::placeholders here (@ the last input argument)

			// Re-normalize the PDF
			//double total_mass = thrust::reduce(thrust::device, GPU_PDF.begin(), GPU_PDF.end());
			//total_mass = disc_X * disc_X * total_mass;
			//thrust::transform(GPU_PDF.begin(), GPU_PDF.end(), GPU_PDF.begin(), 1 / total_mass * _1); // we use the thrust::placeholders here (@ the last input argument)

			*H_PDF = GPU_PDF; // Send back to CPU

		}

		// Store info in cumulative variable
		store_PDFs->insert(store_PDFs->end(), H_PDF->begin(), H_PDF->end());
		j++;


		auto end_2 = std::chrono::high_resolution_clock::now();

		std::chrono::duration<float> duration_2 = end_2 - start_2;
		std::cout << "Liouville iteration took " << duration_2.count() << " seconds\n";
		std::cout << "/-------------------------------------------------------------------/\n";
	}

	return 1;
}