//////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////
//														//
//		Written by: Vicente José Bevia Escrig			//
//		Mathematics Ph.D. student (2020-2024) at:		//
//		Instituto de Matemática Multidisciplinar,		//
//		Universitat Politècnica de València, Spain		//
//														//
//////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////

#include "ivpSolver.cuh"

// --------------------------------------------------------- //
// --------------------------------------------------------- //

//--------------------------------------------------------------------------------------------- //
int16_t PDF_EVOLUTION() {

	hipDeviceProp_t prop;
	errorCheck(IntroDisplay(prop));

	std::cout << "You are simulating the " << CASE <<". Log level is (0 = off, 1 = only frame time, 2 = all steps time): " << OUTPUT_INFO << ".\n\n";
	std::cout << "You must choose: \n - FINEST MESH LEVEL \n - FINAL time \n - TIMESTEP";
	std::cout << "\n - Reinitialization Steps \n - SAMPLES PER PARAMETER \n";
	std::cout << "(REMINDER) You can type \"-1\" to  exit the program while typing this information.\n\n";
	// Pre-Simulation Data initialization -------------------------------------------------------------------------------

	ivpSolver::ivpSolver Solver;

	errorCheck(Solver.buildDomain())
	
	errorCheck(Solver.buildTimeVec())

	errorCheck(Solver.buildDistributions())

auto start = std::chrono::high_resolution_clock::now();

	errorCheck(Solver.evolvePDF(prop))

auto end = std::chrono::high_resolution_clock::now();
	std::chrono::duration<float> duration = end - start; // duration

// -------------------------------------------------------------------------------------------- //
// -------------------------------------------------------------------------------------------- //
// -------------------------------------------------------------------------------------------- //
// ---------------------- STORAGE INTO COMPUTER HDD for post-processing ----------------------- //
// -------------------------------------------------------------------------------------------- //
// -------------------------------------------------------------------------------------------- //
// -------------------------------------------------------------------------------------------- //

	errorCheck(Solver.writeFramesToFile(duration.count()))
	gpuError_Check(hipDeviceReset());

	return 0;
}